#include "hip/hip_runtime.h"
#include "common.h"
#define OUT_TILE_DIM 32


__constant__ float mask_c[MASK_DIM][MASK_DIM]; //constant = cannot write to in in the GPU, but can copy it to GPU from the CPU.
__global__ void convolution_kernel(float* input, float* output, unsigned int width, unsigned int height){
    int outputRow = blockDim.y*blockIdx.y + threadIdx.y;
    int outputCol = blockDim.x*blockIdx.x + threadIdx.x;

    //boundry conditions
    if(outputRow < height && outputCol < width){
        float sum = 0.0f; 
        for(int maskRow = 0; maskRow < MASK_DIM; ++maskRow){
            for(int maskCol = 0; maskCol < MASK_DIM; ++maskCol){
                int inputRow = outputRow - MASK_RADIUS + maskRow;
                int inputCol = outputCol - MASK_RADIUS + maskCol;
                if((inputRow < height && inputRow >= 0) && (inputCol < width && inputCol >=  0)){
                    sum += mask_c[maskRow][maskCol]*input[inputRow*width + inputCol];
                }
            }
        }
        output[outputRow*width + outputCol] = sum;
    }

}

void convolution_gpu(float mask[][MASK_DIM], float* input, float* output, unsigned int width, unsigned int height){
    //alloc mem
    float *input_d, *output_d;
    hipMalloc((void**) &input_d, width*height*sizeof(float));
    hipMalloc((void**) &output_d, width*height*sizeof(float));
    hipDeviceSynchronize();

    //copy data to gpu
    hipMemcpy(input_d, input, width*height*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(output_d, output, width*height*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    //copy mask to gpu
    //we can only allocate upto 64KB, input is also constant but it is too large to put in constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(mask_c), mask, MASK_DIM*MASK_DIM*sizeof(float));
    hipDeviceSynchronize();

    //run kernel
    dim3 numThreadsPerBlock(OUT_TILE_DIM, OUT_TILE_DIM, 1);
    dim3 numBlocks((width + OUT_TILE_DIM - 1)/OUT_TILE_DIM, (height + OUT_TILE_DIM - 1)/OUT_TILE_DIM);
    convolution_kernel<<<numBlocks, numThreadsPerBlock>>>(input_d, output_d, width, height);
    hipDeviceSynchronize();

    //copy mem to cpu
    hipMemcpy(output, output_d, width*height*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    //free mem
    hipFree(input_d);
    hipFree(output_d);
    hipDeviceSynchronize();
}