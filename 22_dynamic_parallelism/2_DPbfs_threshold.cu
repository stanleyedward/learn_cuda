#include "hip/hip_runtime.h"
#include "common.h"

__global__ void bfs_child_kernel(CSRGraph csrGraph, unsigned int *level, unsigned int *currFrontier, unsigned int numPrevFrontier, unsigned int *numCurrFrontier, unsigned int currLevel, unsigned int numNeighbours, unsigned int start)
{

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numNeighbours)
    {
        unsigned int edge = start + i;
        unsigned int neighbour = csrGraph.dst[edge];
        // using atomic operations to prevent raiseconditions for reasons above
        if (atomicCAS(&level[neighbour], UINT_MAX, currLevel) == UINT_MAX) // atomicCAS returns the old value
        {
            unsigned int currFrontierIdx = atomicAdd(numCurrFrontier, 1);
            currFrontier[currFrontierIdx] = neighbour;
        }
    }
}
__global__ void bfs_kernel(CSRGraph csrGraph, unsigned int *level, unsigned int *prevFrontier, unsigned int *currFrontier, unsigned int numPrevFrontier, unsigned int *numCurrFrontier, unsigned int currLevel)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numPrevFrontier)
    {
        unsigned int vertex = prevFrontier[i];
        unsigned int start = csrGraph.srcPtrs[vertex];
        unsigned int numNeighbours = csrGraph.srcPtrs[vertex + 1] - start;
        if (numNeighbours > 1200) // threshold
        {
            unsigned int numThreadsPerBlock = 1024;
            unsigned int numBlocks = (numNeighbours + numThreadsPerBlock - 1) / numThreadsPerBlock;
            bfs_child_kernel<<<numBlocks, numThreadsPerBlock>>>(csrGraph, level, currFrontier, numPrevFrontier, numCurrFrontier, currLevel, numNeighbours, start);
        }
        else // if # of neighbours less than 1200
        {
            for (unsigned int i = 0; i < numNeighbours; ++i)
            {
                unsigned int edge = start + i;
                unsigned int neighbour = csrGraph.dst[edge];
                // using atomic operations to prevent raiseconditions for reasons above
                if (atomicCAS(&level[neighbour], UINT_MAX, currLevel) == UINT_MAX) // atomicCAS returns the old value
                {
                    unsigned int currFrontierIdx = atomicAdd(numCurrFrontier, 1);
                    currFrontier[currFrontierIdx] = neighbour;
                }
            }
        }
    }
}

void bfs_levels(CSRGraph csrGraph, unsigned int *level, unsigned int *prevFrontier, unsigned int *currFrontier, unsigned int *numCurrFrontier)
{

    unsigned int numPrevFrontier = 1;
    unsigned int numThreadsPerBlock = 256;
    hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, csrGraph.numVertices);

    for (unsigned int currLevel = 1; numPrevFrontier > 0; ++currLevel)
    {
        hipMemset(numCurrFrontier, 0, sizeof(unsigned int));
        unsigned int numBlocks = (numPrevFrontier + numThreadsPerBlock - 1) / numThreadsPerBlock;
        bfs_kernel<<<numBlocks, numThreadsPerBlock>>>(csrGraph, level, prevFrontier, currFrontier, numPrevFrontier, numCurrFrontier, currLevel);

        // swap buffers
        unsigned int *tmp = prevFrontier;
        prevFrontier = currFrontier;
        currFrontier = tmp;
        hipMemcpy(&numPrevFrontier, numCurrFrontier, sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
}

void bfs_gpu(CSRGraph csrGraph, unsigned int srcVertex, unsigned int *level)
{
    // alloc mem
    CSRGraph csrGraph_d;
    csrGraph_d.numVertices = csrGraph.numVertices;
    csrGraph_d.numEdges = csrGraph.numEdges;
    hipMalloc((void **)&csrGraph_d.srcPtrs, (csrGraph_d.numVertices + 1) * sizeof(unsigned int));
    hipMalloc((void **)&csrGraph_d.dst, csrGraph_d.numEdges * sizeof(unsigned int));

    unsigned int *level_d;
    hipMalloc((void **)&level_d, csrGraph_d.numVertices * sizeof(unsigned int));

    // for frontier based approach
    unsigned int *buffer1_d;
    unsigned int *buffer2_d;
    unsigned int *numCurrenFrontier_d;
    hipMalloc((void **)&buffer1_d, csrGraph_d.numVertices * sizeof(unsigned int));
    hipMalloc((void **)&buffer2_d, csrGraph_d.numVertices * sizeof(unsigned int));
    hipMalloc((void **)&numCurrenFrontier_d, sizeof(unsigned int));

    unsigned int *prevFrontier_d = buffer1_d;
    unsigned int *currFrontier_d = buffer2_d;
    hipDeviceSynchronize();

    // copy data to GPU
    hipMemcpy(csrGraph_d.srcPtrs, csrGraph.srcPtrs, (csrGraph_d.numVertices + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrGraph_d.dst, csrGraph.dst, csrGraph_d.numEdges * sizeof(unsigned int), hipMemcpyHostToDevice);
    level[srcVertex] = 0;
    hipMemcpy(level_d, level, csrGraph_d.numVertices * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(prevFrontier_d, &srcVertex, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // run kernel
    bfs_levels(csrGraph, level, prevFrontier_d, currFrontier_d, numCurrenFrontier_d);

    // copy reuslt to CPU
    hipMemcpy(level, level_d, csrGraph.numVertices * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // free mem
    hipFree(csrGraph_d.srcPtrs);
    hipFree(csrGraph_d.dst);
    hipFree(level_d);
    hipFree(buffer1_d);
    hipFree(buffer2_d);
    hipFree(numCurrenFrontier_d);
    hipDeviceSynchronize();
}