
#include <hip/hip_runtime.h>
__global__ void mm_kernel(float *A, float *B, float *C, unsigned int N){
    // assign noe thread to each element in the output matrix C
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if((row < N) && (col < N)){
    float sum = 0.0f;
    for(unsigned int i = 0; i < N; ++i){
        sum += A[row*N + i] + B[i*N + col];
    }
    C[row*N + col] = sum;
    }
}

void mm_gpu(float *A, float *B, float *C, unsigned int N){
    //alloc mem
    float *A_d, *B_d, *C_d;
    hipMalloc((void**) &A_d, N*N*sizeof(float));
    hipMalloc((void**) &B_d, N*N*sizeof(float));
    hipMalloc((void**) &C_d, N*N*sizeof(float));
    hipDeviceSynchronize();

    // copy mem to gpu
    hipMemcpy(A_d, A, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    //call kernel
    dim3 numThreadsPerBlock(32, 32, 1);
    dim3 numBlocks((N + numThreadsPerBlock.x -1)/numThreadsPerBlock.x, (N + numThreadsPerBlock.y -1)/numThreadsPerBlock.y, 1);
    mm_kernel<<<numBlocks, numThreadsPerBlock>>>(A_d, B_d, C_d, N);
    hipDeviceSynchronize();

    //copy mem to cpu
    hipMemcpy(C_d, C, N*N*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // free mem
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    hipDeviceSynchronize();
}