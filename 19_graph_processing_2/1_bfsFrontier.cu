#include "hip/hip_runtime.h"
#include "common.h"
__global__ void bfs_kernel(CSRGraph csrGraph, unsigned int *level, unsigned int *prevFrontier, unsigned int *currFrontier, unsigned int numPrevFrontier, unsigned int *numCurrFrontier, unsigned int currLevel)
{
}

void bfs_gpu(CSRGraph csrGraph, unsigned int srcVertex, unsigned int *level)
{
    // alloc mem
    CSRGraph csrGraph_d;
    csrGraph_d.numVertices = csrGraph.numVertices;
    csrGraph_d.numEdges = csrGraph.numEdges;
    hipMalloc((void **)&csrGraph_d.srcPtrs, (csrGraph_d.numVertices + 1) * sizeof(unsigned int));
    hipMalloc((void **)&csrGraph_d.dst, csrGraph_d.numEdges * sizeof(unsigned int));

    unsigned int *level_d;
    hipMalloc((void **)&level_d, csrGraph_d.numVertices * sizeof(unsigned int));

    // for frontier based approach
    unsigned int *buffer1_d;
    unsigned int *buffer2_d;
    unsigned int *numCurrenFrontier_d;
    hipMalloc((void **)&buffer1_d, csrGraph_d.numVertices * sizeof(unsigned int));
    hipMalloc((void **)&buffer2_d, csrGraph_d.numVertices * sizeof(unsigned int));
    hipMalloc((void **)&numCurrenFrontier_d, sizeof(unsigned int));

    unsigned int *prevFrontier_d = buffer1_d;
    unsigned int *currFrontier_d = buffer2_d;
    hipDeviceSynchronize();

    // copy data to GPU
    hipMemcpy(csrGraph_d.srcPtrs, csrGraph.srcPtrs, (csrGraph_d.numVertices + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrGraph_d.dst, csrGraph.dst, csrGraph_d.numEdges * sizeof(unsigned int), hipMemcpyHostToDevice);
    level[srcVertex] = 0;
    hipMemcpy(level_d, level, csrGraph_d.numVertices * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(prevFrontier_d, &srcVertex, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // run kernel
    unsigned int numPrevFrontier = 1;
    unsigned int numThreadsPerBlock = 256;
    for (unsigned int currLevel = 1; numPrevFrontier > 0; ++currLevel)
    {
        hipMemset(numCurrenFrontier_d, 0, sizeof(unsigned int));
        unsigned int numBlocks = (numPrevFrontier + numThreadsPerBlock - 1) / numThreadsPerBlock;
        bfs_kernel<<<numBlocks, numThreadsPerBlock>>>(csrGraph_d, level_d, prevFrontier_d, currFrontier_d, numPrevFrontier, numCurrenFrontier_d, currLevel);

        // swap buffers
        unsigned int *tmp = prevFrontier_d;
        prevFrontier_d = currFrontier_d;
        currFrontier_d = tmp;
        hipMemcpy(&numPrevFrontier, numCurrenFrontier_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();

    // copy reuslt to CPU
    hipMemcpy(level, level_d, csrGraph.numVertices * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // free mem
    hipFree(csrGraph_d.srcPtrs);
    hipFree(csrGraph_d.dst);
    hipFree(level_d);
    hipFree(buffer1_d);
    hipFree(buffer2_d);
    hipFree(numCurrenFrontier_d);
    hipDeviceSynchronize();
}