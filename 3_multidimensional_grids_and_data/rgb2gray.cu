
#include <hip/hip_runtime.h>
__global__ void rgb2gray_kernel(unsigned char *red, unsigned char *green, unsigned char *blue, unsigned char *gray, unsigned int width, unsigned int height){
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    // since R G and B in C memory is stored in 1d and using row major order, to get 1D index from 2D
    unsigned int index = row*width + col;
    if (row < height && col < width){
    gray[index] = red[index]*3/10 + green[index]*6/10 + blue[index]*1/10; 
    }
}


void rgb2gray_gpu(unsigned char *red, unsigned char *green, unsigned char *blue, unsigned char *gray, unsigned int width, unsigned int height){
    // Allocate GPU memory
    unsigned char *red_d, *green_d, *blue_d, *gray_d;
    hipMalloc((void**) &red_d, width*height*sizeof(unsigned char));
    hipMalloc((void**) &green_d, width*height*sizeof(unsigned char));
    hipMalloc((void**) &blue_d, width*height*sizeof(unsigned char));
    hipMalloc((void**) &gray_d, width*height*sizeof(unsigned char));

    hipDeviceSynchronize();

    //cpoy data to GPU
    hipMemcpy(red_d, red, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(green_d, green, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(blue_d, blue, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);

    hipDeviceSynchronize();


    // Call kernel
    dim3 numThreadsPerBlock(32, 32, 1);// an int dtype with 3 dim x, y and z
    dim3 numBlocks((width + numThreadsPerBlock.x -1)/numThreadsPerBlock.x, (height + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y, 1);
    rgb2gray_kernel <<<numBlocks, numThreadsPerBlock>>> (red_d, green_d, blue_d, gray_d, width, height);

    hipDeviceSynchronize();

    // copy data from the GPU to CPU
    hipMemcpy(gray, gray_d, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); 

    // free GPU memory
    hipFree(red_d);
    hipFree(green_d);
    hipFree(blue_d);
    hipFree(gray_d);
    hipDeviceSynchronize();
}

