#include "hip/hip_runtime.h"
#include "common.h"
__global__ void bfs_kernel(COOGraph cooGraph, unsigned int *level, unsigned int *newVertexVisited, unsigned int currLevel)
{
    unsigned int edge = blockIdx.x * blockDim.x + threadIdx.x;
    if (edge < cooGraph.numEdges)
    {
        unsigned int vertex = cooGraph.src[edge];
        unsigned int neighbour = cooGraph.dst[edge];
        if (level[vertex] == currLevel - 1 && level[neighbour] == UINT_MAX)
        {
            level[neighbour] == currLevel;
            *newVertexVisited = 1;
        }
    }
}

void bfs_gpu(COOGraph cooGraph, unsigned int srcVertex, unsigned int *level)
{
    // alloc mem
    COOGraph cooGraph_d;
    cooGraph_d.numVertices = cooGraph.numVertices;
    cooGraph_d.numEdges = cooGraph.numEdges;
    hipMalloc((void **)&cooGraph_d.src, cooGraph_d.numEdges * sizeof(unsigned int));
    hipMalloc((void **)&cooGraph_d.dst, cooGraph_d.numEdges * sizeof(unsigned int));

    unsigned int *level_d;
    hipMalloc((void **)&level_d, cooGraph_d.numVertices * sizeof(unsigned int));
    unsigned int *newVertexVisited_d;
    hipMalloc((void **)&newVertexVisited_d, sizeof(unsigned int));
    hipDeviceSynchronize();

    // copy data to GPU
    hipMemcpy(cooGraph_d.src, cooGraph.src, cooGraph_d.numEdges * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(cooGraph_d.dst, cooGraph.dst, cooGraph_d.numEdges * sizeof(unsigned int), hipMemcpyHostToDevice);
    level[srcVertex] = 0;
    hipMemcpy(level_d, level, cooGraph_d.numVertices * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // run kernel
    unsigned int numThreadsPerBlock = 256;
    unsigned int numBlocks = (cooGraph_d.numEdges + numThreadsPerBlock - 1) / numThreadsPerBlock;
    unsigned int newVertexVisited = 1;
    for (unsigned int currLevel = 1; newVertexVisited; ++currLevel)
    {
        newVertexVisited = 0;
        hipMemcpy(newVertexVisited_d, &newVertexVisited, sizeof(unsigned int), hipMemcpyHostToDevice);
        bfs_kernel<<<numBlocks, numThreadsPerBlock>>>(cooGraph_d, level_d, newVertexVisited_d, currLevel);
        hipMemcpy(&newVertexVisited, newVertexVisited_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();

    // copy reuslt to CPU
    hipMemcpy(level, level_d, cooGraph.numVertices * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // free mem
    hipFree(cooGraph_d.src);
    hipFree(cooGraph_d.dst);
    hipFree(level_d);
    hipFree(newVertexVisited_d);
}