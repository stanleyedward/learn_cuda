#include "hip/hip_runtime.h"
#include "common.h"

__global__ void spmv_coo_kernel(COOMatrix cooMatrix, float *inVector, float *outVector)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < cooMatrix.numNonzeros)
    {
        unsigned int row = cooMatrix.rowIds[i];
        unsigned int col = cooMatrix.colIds[i];
        float value = cooMatrix.values[i];

        atomicAdd(&outVector[row], inVector[col]*value);
    }
}

void spmv_coo_gpu(COOMatrix cooMatrix, float *inVector, float *outVector)
{
    // alloc mem
    COOMatrix cooMatrix_d;
    cooMatrix_d.numRows = cooMatrix.numRows;
    cooMatrix_d.numCols = cooMatrix.numCols;
    cooMatrix_d.numNonzeros = cooMatrix.numNonzeros;
    hipMalloc((void **)&cooMatrix_d.rowIds, cooMatrix_d.numNonzeros * sizeof(unsigned int));
    hipMalloc((void **)&cooMatrix_d.colIds, cooMatrix_d.numNonzeros * sizeof(unsigned int));
    hipMalloc((void **)&cooMatrix_d.values, cooMatrix_d.numNonzeros * sizeof(float));
    float *inVector_d;
    hipMalloc((void **)&inVector_d, cooMatrix_d.numCols * sizeof(float));
    float *outVector_d;
    hipMalloc((void **)&outVector_d, cooMatrix_d.numRows * sizeof(float));
    hipDeviceSynchronize();

    // cpy data to gpu
    hipMemcpy(cooMatrix_d.rowIds, cooMatrix.rowIds, cooMatrix_d.numNonzeros * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(cooMatrix_d.colIds, cooMatrix.colIds, cooMatrix_d.numNonzeros * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(cooMatrix_d.values, cooMatrix.values, cooMatrix_d.numNonzeros * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(inVector_d, inVector, cooMatrix.numCols * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(outVector_d, 0, cooMatrix.numRows * sizeof(float));
    hipDeviceSynchronize();

    // call the kernel
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (cooMatrix_d.numNonzeros + numThreadsPerBlock - 1) / numThreadsPerBlock;
    spmv_coo_kernel<<<numBlocks, numThreadsPerBlock>>>(cooMatrix_d, inVector_d, outVector_d);
    hipDeviceSynchronize();

    // copy data to CPU
    hipMemcpy(outVector, outVector_d, cooMatrix.numRows * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Free GPU memory
    hipFree(cooMatrix_d.rowIds);
    hipFree(cooMatrix_d.colIds);
    hipFree(cooMatrix_d.values);
    hipFree(inVector_d);
    hipFree(outVector_d);
    hipDeviceSynchronize();
}