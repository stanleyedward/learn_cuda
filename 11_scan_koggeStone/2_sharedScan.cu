
#include <hip/hip_runtime.h>
#define BLOCK_DIM 1024
__global__ void scan_kernel(float *input, float *output, float *partialSums, unsigned int N)
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ float buffer_s[BLOCK_DIM];
    // to access the shared memory buffer we use local idx not global idx
    buffer_s[threadIdx.x] = input[i];
    __syncthreads();
    for (unsigned int stride = 1; stride <= BLOCK_DIM / 2; stride *= 2)
    {
        float v;
        //sync after read
        if (threadIdx.x >= stride)
        {
            v = buffer_s[threadIdx.x - stride];
        }
        __syncthreads();
        //sync after write
        if (threadIdx.x >= stride)
        {
            buffer_s[threadIdx.x] += v;
        }
        __syncthreads();
    }
    if (threadIdx.x == BLOCK_DIM - 1)
    {
        partialSums[blockIdx.x] = buffer_s[threadIdx.x];
    }
    output[i] = buffer_s[threadIdx.x];
}

__global__ void add_kernel(float *output, float *partialSums, unsigned int N)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (blockIdx.x > 0)
    {
        output[i] += partialSums[blockIdx.x - 1];
    }
}

void scan_gpu_d(float *input_d, float *output_d, unsigned int N)
{
    // configurations
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    const unsigned int numElementsPerBlock = numThreadsPerBlock;
    const unsigned int numBlocks = (N + numElementsPerBlock - 1) / numElementsPerBlock;

    // alloc partial sums
    float *partialSums_d;
    hipMalloc((void **)&partialSums_d, numBlocks * sizeof(float));
    hipDeviceSynchronize();

    // call kernel
    scan_kernel<<<numBlocks, numThreadsPerBlock>>>(input_d, output_d, partialSums_d, N);
    hipDeviceSynchronize();

    // scan partial sums then add
    if (numBlocks > 1)
    {
        // scan partial sums
        scan_gpu_d(partialSums_d, partialSums_d, numBlocks);

        // add scanned sums
        add_kernel<<<numBlocks, numThreadsPerBlock>>>(output_d, partialSums_d, N);
    }

    // free memory
    hipFree(partialSums_d);
    hipDeviceSynchronize();
}