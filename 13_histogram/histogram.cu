#include "hip/hip_runtime.h"
#include "common.h"
__global__ void histogram_kernel(unsigned char *image, unsigned int *bins, unsigned int width, unsigned int height)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width * height){
        unsigned char b = image[i];
        atomicAdd(&bins[b], 1);
    }
}

void histogram_gpu(unsigned char *image, unsigned int *bins, unsigned int width, unsigned int height)
{
    // alloc mem
    unsigned char *image_d;
    unsigned int *bins_d;
    hipMalloc((void **)&image_d, width * height * sizeof(unsigned char));
    hipMalloc((void **)&bins_d, NUM_BINS * sizeof(unsigned int));
    hipDeviceSynchronize();

    // copy data to gpu
    hipMemcpy(image_d, image, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemset(bins_d, 0, NUM_BINS * sizeof(unsigned int)); // bins start at 0 before incementings
    hipDeviceSynchronize();

    // call kernel
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (width * height + numThreadsPerBlock - 1) / numThreadsPerBlock;
    histogram_kernel<<<numBlocks, numThreadsPerBlock>>>(image_d, bins_d, width, height);
    hipDeviceSynchronize();

    // copy data to CPU
    hipMemcpy(bins, bins_d, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Free GPU memory
    hipFree(image_d);
    hipFree(bins_d);
    hipDeviceSynchronize();