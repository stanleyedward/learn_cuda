
#include <hip/hip_runtime.h>
#define BLOCK_DIM 1024

__global__ void reduce_kernel(float *input, float *partialSums, unsigned int N)
{
    unsigned int segment = blockIdx.x * blockDim.x * 2;
    unsigned int i = segment + threadIdx.x;
    for (unsigned int stride = BLOCK_DIM; stride > 0; stride /= 2)
    {
        if (threadIdx.x < stride)
        {
            input[i] += input[i + stride];
        }
        __syncthreads();
    }
    // every thread will have a partial sum at threadIdx 0 in the block, therefore have thread 0 store its result in the partial sum array.
    if (threadIdx.x == 0)
    {
        partialSums[blockIdx.x] = input[i];
    }
}

float reduce_gpu(float *input, unsigned int N)
{
    // alloc amem
    float *input_d;
    hipMalloc((void **)&input_d, N * sizeof(float));
    hipDeviceSynchronize();

    // copy data to GPU
    hipMemcpy(input_d, input, N * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // alloc partial sums
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    const unsigned int numElementsPerBlock = 2 * numThreadsPerBlock;
    const unsigned int numBlocks = (N + numElementsPerBlock - 1) / numElementsPerBlock;
    float *partialSums = (float *)malloc(numBlocks * sizeof(float));
    float *partialSums_d;
    hipMalloc((void **)&partialSums_d, numBlocks * sizeof(float));
    hipDeviceSynchronize();

    // call kernel
    reduce_kernel<<<numBlocks, numThreadsPerBlock>>>(input_d, partialSums_d, N);
    hipDeviceSynchronize();

    // copy data from GPU
    hipMemcpy(partialSums, partialSums_d, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // reduce partial sums on CPU
    float sum = 0.0f;
    for (unsigned int i = 0; i < numBlocks; ++i)
    {
        sum += partialSums[i];
    }

    // free mem
    hipFree(input_d);
    hipFree(partialSums_d);
    free(partialSums);
    hipDeviceSynchronize();
}