
#include <hip/hip_runtime.h>
#define BLOCK_DIDM 1024

__global__ void reduce_kernel(float* input, float* partialSums, unsigned int N){


}

float reduce_gpu(float* input, unsigned int N){
    //alloc amem
    float *input_d;
    hipMalloc((void**) &input_d, N*sizeof(float));
    hipDeviceSynchronize();

    //copy data to GPU
    hipMemcpy(input_d, input, N*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    //alloc partial sums
    const unsigned int numThreadsPerBlock = BLOCK_DIDM;
    const unsigned int numElementsPerBlock = 2*numThreadsPerBlock;
    const unsigned int numBlocks = (N + numElementsPerBlock -1)/numElementsPerBlock;
    float* partialSums = (float*)malloc(numBlocks*sizeof(float));
    float* partialSums_d;
    hipMalloc((void**)&partialSums_d, numBlocks*sizeof(float));
    hipDeviceSynchronize();

    //call kernel
    reduce_kernel <<<numBlocks, numThreadsPerBlock>>> (input_d, partialSums_d, N);
    hipDeviceSynchronize();

    //copy data from GPU
    hipMemcpy(partialSums, partialSums_d, numBlocks*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // reduce partial sums on CPU
    float sum = 0.0f;
    for(unsigned int i = 0; i < numBlocks; ++i){
        sum += partialSums[i];
    }

    //free mem
    hipFree(input_d);
    hipFree(partialSums_d);
    free(partialSums);
    hipDeviceSynchronize();

}