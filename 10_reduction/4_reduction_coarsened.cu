
#include <hip/hip_runtime.h>
 #define BLOCK_DIM 1024
#define COARSE_FACTOR 4

__global__ void reduce_kernel(float *input, float *partialSums, unsigned int N)
{
    unsigned int segment = blockIdx.x * blockDim.x * 2 * COARSE_FACTOR;
    unsigned int i = segment + threadIdx.x;

    __shared__ float input_s[BLOCK_DIM];
    float sum = 0.0f;
    for(unsigned int tile = 0;tile < COARSE_FACTOR*2; ++tile){
        // only add boundary conditions here
        sum += input[i + tile*BLOCK_DIM];
    }
    input_s[threadIdx.x] = sum;
    __syncthreads();
    for (unsigned int stride = BLOCK_DIM/2; stride > 0; stride /= 2)
    {
        if (threadIdx.x < stride)
        {
            input_s[threadIdx.x] += input[threadIdx.x + stride];
        }
        __syncthreads();
    }
    // every thread will have a partial sum at threadIdx 0 in the block, therefore have thread 0 store its result in the partial sum array.
    if (threadIdx.x == 0)
    {
        partialSums[blockIdx.x] = input_s[i];
    }
}

float reduce_gpu(float *input, unsigned int N)
{
    // alloc amem
    float *input_d;
    hipMalloc((void **)&input_d, N * sizeof(float));
    hipDeviceSynchronize();

    // copy data to GPU
    hipMemcpy(input_d, input, N * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // alloc partial sums
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    // since each thread block is resposible for more data in coarsening 
    const unsigned int numElementsPerBlock = 2 * numThreadsPerBlock * COARSE_FACTOR;
    const unsigned int numBlocks = (N + numElementsPerBlock - 1) / numElementsPerBlock;
    float *partialSums = (float *)malloc(numBlocks * sizeof(float));
    float *partialSums_d;
    hipMalloc((void **)&partialSums_d, numBlocks * sizeof(float));
    hipDeviceSynchronize();

    // call kernel
    reduce_kernel<<<numBlocks, numThreadsPerBlock>>>(input_d, partialSums_d, N);
    hipDeviceSynchronize();

    // copy data from GPU
    hipMemcpy(partialSums, partialSums_d, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // reduce partial sums on CPU
    float sum = 0.0f;
    for (unsigned int i = 0; i < numBlocks; ++i)
    {
        sum += partialSums[i];
    }

    // free mem
    hipFree(input_d);
    hipFree(partialSums_d);
    free(partialSums);
    hipDeviceSynchronize();
}